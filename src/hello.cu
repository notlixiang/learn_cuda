#include "hip/hip_runtime.h"
/*
 * @Date: 2020-09-11 14:42:51
 * @LastEditTime: 2020-09-21 16:17:03
 * @LastEditors: Li Xiang
 * @Description: learn_cuda
 * @FilePath: /src/learn_cuda/src/hello.cpp
 */

#include <ros/ros.h>
#include <iostream>
#include <math.h>
#include <chrono>

using namespace std;
using namespace std::chrono;

// __global__ functions, or "kernels", execute on the device
__global__ void hello_kernel(void)
{
  printf("Hello, world from the device!\n");
}

int main(void)
{
  // greet from the host
  printf("Hello, world from the host!\n");

  // launch a kernel with a single thread to greet from the device
  hello_kernel<<<1,1>>>();

  // wait for the device to finish so that we see the message
  hipDeviceSynchronize();

  return 0;
}